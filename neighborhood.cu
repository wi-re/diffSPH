#include "hip/hip_runtime.h"
#include "neighborhood.h"
template<std::size_t dim>
__global__ void buildNeighborhoodCudaDispatcher(int32_t numParticles,
                                                cptr_t<int32_t, 1> neighborOffsets, ptr_t<int32_t, 1> neighborList_i, ptr_t<int32_t, 1> neighborList_j,
                                                cptr_t<float, 2> queryPositions, cptr_t<float, 1> querySupport, int searchRange,
                                                cptr_t<float, 2> sortedPositions, cptr_t<float, 1> sortedSupport,
                                                cptr_t<int32_t, 2> hashTable, int hashMapLength,
                                                cptr_t<int64_t, 2> cellTable, cptr_t<int32_t, 1> numCells,
                                                cptr_t<int32_t, 2> offsets, float hCell, cptr_t<float, 1> minDomain, cptr_t<float, 1> maxDomain, cptr_t<int32_t, 1> periodicity,
                                                supportMode searchMode) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        buildNeighborhood<dim>(i, neighborOffsets, neighborList_i, neighborList_j, queryPositions, querySupport, searchRange, sortedPositions, sortedSupport, hashTable, hashMapLength, cellTable, numCells, offsets, hCell, minDomain, maxDomain, periodicity, searchMode);
    }
}
template<std::size_t dim>
__global__ void countNeighborsForParticleCudaDispatcher(int32_t numParticles,
                                                        ptr_t<int32_t, 1> neighborCounters,
                                                        cptr_t<float, 2> queryPositions, cptr_t<float, 1> querySupport, int searchRange,
                                                        cptr_t<float, 2> sortedPositions, cptr_t<float, 1> sortedSupport,
                                                        cptr_t<int32_t, 2> hashTable, int hashMapLength,
                                                        cptr_t<int64_t, 2> cellTable, cptr_t<int32_t, 1> numCellsVec,
                                                        cptr_t<int32_t, 2> offsets,
                                                        float hCell, cptr_t<float, 1> minDomain, cptr_t<float, 1> maxDomain, cptr_t<int32_t, 1> periodicity,
                                                        supportMode searchMode) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        countNeighborsForParticle<dim>(i, neighborCounters, queryPositions, querySupport, searchRange, sortedPositions, sortedSupport, hashTable, hashMapLength, cellTable, numCellsVec, offsets, hCell, minDomain, maxDomain, periodicity, searchMode);
    }
}

#include <hip/hip_runtime.h>
void cuda_error_check() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}

template<typename Func, typename... Ts>
void launchKernel(Func kernel, int numParticles, Ts&&... args) {
    int blockSize;  // Number of threads per block
    int minGridSize;  // Minimum number of blocks required for the kernel
    int gridSize;  // Number of blocks to use

    // Compute the maximum potential block size for the kernel
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel);
    // cuda_error_check();
    gridSize = (numParticles + blockSize - 1) / blockSize;

    kernel<<<gridSize, blockSize>>>(numParticles, std::forward<Ts>(args)...);
    // cuda_error_check();
}


void buildNeighborhoodCuda(torch::Tensor neighborOffsets, torch::Tensor neighborList_i, torch::Tensor neighborList_j,
    torch::Tensor queryPositions, torch::Tensor querySupport, int searchRange,
    torch::Tensor sortedPositions, torch::Tensor sortedSupport,
    torch::Tensor hashTable, int hashMapLength,
    torch::Tensor cellTable, torch::Tensor numCells,
    torch::Tensor offsets, float hCell, torch::Tensor minDomain, torch::Tensor maxDomain, torch::Tensor periodicity,
    supportMode searchMode) {
    int32_t numParticles = queryPositions.size(0);
    
    int32_t threads = 1024;
    int32_t blocks = (int32_t)floor(numParticles / threads) + (numParticles % threads == 0 ? 0 : 1);

#define args numParticles, \
neighborOffsets.packed_accessor32<int32_t,1, traits>(), neighborList_i.packed_accessor32<int32_t,1, traits>(), neighborList_j.packed_accessor32<int32_t,1, traits>(), \
queryPositions.packed_accessor32<float, 2, traits>(), querySupport.packed_accessor32<float,1, traits>(), searchRange, \
sortedPositions.packed_accessor32<float, 2, traits>(), sortedSupport.packed_accessor32<float,1, traits>(), \
hashTable.packed_accessor32<int32_t,2, traits>(), hashMapLength, \
cellTable.packed_accessor32<int64_t,2, traits>(), numCells.packed_accessor32<int32_t,1, traits>(), \
offsets.packed_accessor32<int32_t,2, traits>(), \
hCell, minDomain.packed_accessor32<float,1, traits>(), maxDomain.packed_accessor32<float,1, traits>(), periodicity.packed_accessor32<int32_t,1, traits>(), searchMode

    int32_t dim = queryPositions.size(1);
    if(dim == 1)
        launchKernel(buildNeighborhoodCudaDispatcher<1>, args);
        // buildNeighborhoodCudaDispatcher<1><<<blocks, threads>>>(args);
    else if(dim == 2)
        launchKernel(buildNeighborhoodCudaDispatcher<2>, args);
        // buildNeighborhoodCudaDispatcher<2><<<blocks, threads>>>(args);
    else if(dim == 3)
        launchKernel(buildNeighborhoodCudaDispatcher<3>, args);
        // buildNeighborhoodCudaDispatcher<3><<<blocks, threads>>>(args);
    else throw std::runtime_error("Unsupported dimensionality");

    // cuda_error_check();

#undef args
}

void countNeighborsForParticleCuda(
    torch::Tensor neighborCounters, 
    torch::Tensor queryPositions, torch::Tensor querySupport, int searchRange, 
    torch::Tensor sortedPositions, torch::Tensor sortedSupport,
    torch::Tensor hashTable, int hashMapLength,
    torch::Tensor cellTable, torch::Tensor numCellsVec, 
    torch::Tensor offsets,
    float hCell, torch::Tensor minDomain, torch::Tensor maxDomain, torch::Tensor periodicity,
    supportMode searchMode) {
    int32_t numParticles = queryPositions.size(0);
    int32_t threads = 32;
    int32_t blocks = (int32_t)floor(numParticles / threads) + (numParticles % threads == 0 ? 0 : 1);

#define args \
        numParticles, \
        neighborCounters.packed_accessor32<int32_t,1, traits>(), \
        queryPositions.packed_accessor32<float,2, traits>(), querySupport.packed_accessor32<float,1, traits>(), searchRange, \
        sortedPositions.packed_accessor32<float,2, traits>(), sortedSupport.packed_accessor32<float,1, traits>(), \
        hashTable.packed_accessor32<int32_t,2, traits>(), hashMapLength, \
        cellTable.packed_accessor32<int64_t,2, traits>(), numCellsVec.packed_accessor32<int32_t,1, traits>(), \
        offsets.packed_accessor32<int32_t,2, traits>(), \
        hCell, minDomain.packed_accessor32<float, 1, traits>(), maxDomain.packed_accessor32<float, 1, traits>(), periodicity.packed_accessor32<int32_t, 1, traits>(), searchMode

    int32_t dim = queryPositions.size(1);
    // std::cout << "dim: " << dim << std::endl;
    if (dim == 1)
        launchKernel(countNeighborsForParticleCudaDispatcher<1>, args);
        // countNeighborsForParticleCudaDispatcher<1><<<blocks, threads>>>(args);
    else if (dim == 2)
        launchKernel(countNeighborsForParticleCudaDispatcher<2>, args);
        // countNeighborsForParticleCudaDispatcher<2><<<blocks, threads>>>(args);
    else if (dim == 3)
        launchKernel(countNeighborsForParticleCudaDispatcher<3>, args);
        // countNeighborsForParticleCudaDispatcher<3><<<blocks, threads>>>(args);
    else throw std::runtime_error("Unsupported dimensionality");

#undef args
}
